#include <stdio.h>
#include <papi.h>
#include "gpu_work.h"

#define COMP_NAME "cuda_pw"

#define NUM_METRICS 1
const char *test_metrics[] = {
    COMP_NAME ":::smsp__warps_launched.sum:device=0",
};

void multi_reset(long long *values)
{
    int EventSet = PAPI_NULL;
    int res, i, j;
    hipCtx_t ctx;
    res = PAPI_create_eventset(&EventSet);
    if (res != PAPI_OK) {
        fprintf(stderr, "Failed to create eventset.\n");
    }

    res = hipCtxCreate(&ctx, 0, 0);
    if (res != hipSuccess) {
        fprintf(stderr, "cuda error: failed to create cuda context.\n");
    }

    for (i=0; i<NUM_METRICS; i++) {
        res = PAPI_add_named_event(EventSet, test_metrics[i]);
        if (res != PAPI_OK) {
            fprintf(stderr, "Failed to add event %s\n", test_metrics[i]);
        }
    }

    res = PAPI_start(EventSet);
    if (res != PAPI_OK) {
        fprintf(stderr, "PAPI_start error.\n");
    }
    for (i=0; i<10; i++) {
        VectorAddSubtract(100000);
        res = PAPI_read(EventSet, values);
        if (res != PAPI_OK) {
            fprintf(stderr, "PAPI_read error.\n");
        }
        printf("Measured values iter %d\n", i);
        for (j=0; j<NUM_METRICS; j++) {
            printf("%s\t\t%lld\n", test_metrics[j], values[j]);
        }
        res = PAPI_reset(EventSet);
    }
    res = PAPI_stop(EventSet, values);
    if (res != PAPI_OK) {
        fprintf(stderr, "PAPI_stop error.\n");
    }
    res = PAPI_cleanup_eventset(EventSet);
    if (res != PAPI_OK) {
        fprintf(stderr, "PAPI_cleanup_eventset error.\n");
    }
    res = hipCtxDestroy(ctx);
    if (res != hipSuccess) {
        fprintf(stderr, "cude error: failed to destroy context.\n");
    }
}

void multi_read(long long *values)
{
    int EventSet = PAPI_NULL;
    int res, i, j;
    hipCtx_t ctx;
    res = PAPI_create_eventset(&EventSet);
    if (res != PAPI_OK) {
        fprintf(stderr, "Failed to create eventset.\n");
    }

    res = hipCtxCreate(&ctx, 0, 0);
    if (res != hipSuccess) {
        fprintf(stderr, "cuda error: failed to create cuda context.\n");
    }

    for (i=0; i<NUM_METRICS; i++) {
        res = PAPI_add_named_event(EventSet, test_metrics[i]);
        if (res != PAPI_OK) {
            fprintf(stderr, "Failed to add event %s\n", test_metrics[i]);
        }
    }

    res = PAPI_start(EventSet);
    if (res != PAPI_OK) {
        fprintf(stderr, "PAPI_start error.\n");
    }
    for (i=0; i<10; i++) {
        VectorAddSubtract(100000);
        res = PAPI_read(EventSet, values);
        if (res != PAPI_OK) {
            fprintf(stderr, "PAPI_read error.\n");
        }
        printf("Measured values iter %d\n", i);
        for (j=0; j<NUM_METRICS; j++) {
            printf("%s\t\t%lld\n", test_metrics[j], values[j]);
        }
    }
    res = PAPI_stop(EventSet, values);
    if (res != PAPI_OK) {
        fprintf(stderr, "PAPI_stop error.\n");
    }
    res = PAPI_cleanup_eventset(EventSet);
    if (res != PAPI_OK) {
        fprintf(stderr, "PAPI_cleanup_eventset error.\n");
    }
    res = hipCtxDestroy(ctx);
    if (res != hipSuccess) {
        fprintf(stderr, "cude error: failed to destroy context.\n");
    }
}

void single_read(long long *values)
{
    int EventSet = PAPI_NULL;
    int res, i, j;
    hipCtx_t ctx;
    res = PAPI_create_eventset(&EventSet);
    if (res != PAPI_OK) {
        fprintf(stderr, "PAPI_create_eventset error.\n");
    }
    res = hipCtxCreate(&ctx, 0, 0);
    if (res != hipSuccess) {
        fprintf(stderr, "cuda error: failed to create cuda context.\n");
    }
    for (i=0; i<NUM_METRICS; i++) {
        res = PAPI_add_named_event(EventSet, test_metrics[i]);
        if (res != PAPI_OK) {
            fprintf(stderr, "Failed to add event %s\n", test_metrics[i]);
        }
    }

    res = PAPI_start(EventSet);
    if (res != PAPI_OK) {
        fprintf(stderr, "PAPI_start error.\n");
    }
    for (i=0; i<10; i++) {
        VectorAddSubtract(100000);
    }
    res = PAPI_stop(EventSet, values);
    if (res != PAPI_OK) {
        fprintf(stderr, "PAPI_stop error.\n");
    }
    printf("Measured values from single read\n");
    for (j=0; j<NUM_METRICS; j++) {
        printf("%s\t\t%lld\n", test_metrics[j], values[j]);
    }
    res = PAPI_cleanup_eventset(EventSet);
    if (res != PAPI_OK) {
        fprintf(stderr, "PAPI_cleanup_eventset error.\n");
    }
    res = hipCtxDestroy(ctx);
    if (res != hipSuccess) {
        fprintf(stderr, "cuda error: failed to destroy cuda context.\n");
    }
}

int main()
{
    int res;
    res = hipInit(0);

    res = PAPI_library_init(PAPI_VER_CURRENT);
    if (res != PAPI_VER_CURRENT) {
        fprintf(stderr, "Failed to initialize PAPI.\n");
    }
    res = PAPI_get_component_index(COMP_NAME);
    if (res < 0) {
        fprintf(stderr, "PAPI not configured with '" COMP_NAME "' component!");
        exit(-1);
    }
    long long values_multi_reset[NUM_METRICS];
    long long values_multi_read[NUM_METRICS];
    long long values_single_read[NUM_METRICS];

    printf("Running multi_reset.\n");
    multi_reset(values_multi_reset);
    printf("\nRunning multi_read.\n");
    multi_read(values_multi_read);
    printf("\nRunning single_read.\n");
    single_read(values_single_read);

    int i;
    printf("Final measured values\nEvent_name\t\t\t\t\t\tMulti_read\tsingle_read\n");
    for (i=0; i<NUM_METRICS; i++) {
        printf("%s\t\t\t%lld\t\t%lld\n", test_metrics[i], values_multi_read[i], values_single_read[i]);
    }
    PAPI_shutdown();
    return 0;
}
