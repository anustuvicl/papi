#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include "gpu_work.h"
#include <papi.h>
#include <papi_test.h>

#define PAPI_CALL(apiFuncCall)                                          \
do {                                                                           \
    int _status = apiFuncCall;                                         \
    if (_status != PAPI_OK) {                                              \
        fprintf(stderr, "%s:%d: error %d: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, _status, #apiFuncCall, PAPI_strerror(_status));\
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define NUM_THREADS 2
// User metrics to profile
#define NUM_METRICS 2
const char *test_metrics[] = {
    "cuda:::smsp__warps_launched.sum",
    "cuda:::dram__bytes_write.sum",
};

int numGPUs;

pthread_t tidarr[NUM_THREADS];
hipCtx_t cuCtx[NUM_THREADS];
pthread_mutex_t lock;

void * thread_gpu(void * idx)
{
    int tid = *((int*) idx);
    int gpuid = tid % numGPUs;
    unsigned long gettid = (unsigned long) pthread_self();
    int retval;

    int EventSet = PAPI_NULL;
    long long values[1];
    PAPI_CALL(PAPI_create_eventset(&EventSet));

    DRIVER_API_CALL(hipCtxSetCurrent(cuCtx[tid]));
    fprintf(stderr, "This is idx %d thread %lu - using GPU %d context %p!\n",
            tid, gettid, gpuid, cuCtx[tid]);

    char tmpEventName[64];
    snprintf(tmpEventName, 64, "%s:device=%d", test_metrics[tid], 0);
    retval = PAPI_add_named_event(EventSet, tmpEventName);
    if (retval != PAPI_OK) {
        fprintf(stderr, "Failed to add event %s\n", tmpEventName);
    }

    retval = PAPI_start(EventSet);
    if (retval == PAPI_ECNFLCT) {
        test_pass(__FILE__);
        return NULL;
    }
    VectorAddSubtract(5000000*(tid+1));  // gpu work

    PAPI_CALL(PAPI_stop(EventSet, values));

    printf("User measured values in thread id %d.\n", tid);
    snprintf(tmpEventName, 64, "%s:device=%d", test_metrics[tid], gpuid);
    printf("%s\t\t%lld\n", tmpEventName, values[0]);
    return NULL;
}

int main()
{
    int rc, i;
    int tid[NUM_THREADS];
    RUNTIME_API_CALL(hipGetDeviceCount(&numGPUs));
    printf("No. of GPUs = %d\n", numGPUs);
    printf("No. of threads to launch = %d\n", NUM_THREADS);

    int retval = PAPI_library_init( PAPI_VER_CURRENT );
    if( retval != PAPI_VER_CURRENT ) {
        fprintf( stderr, "PAPI_library_init failed\n" );
        exit(-1);
    }
    // Point PAPI to function that gets the thread id
    PAPI_CALL(PAPI_thread_init((unsigned long (*)(void)) pthread_self));

    // Launch the threads
    for(i = 0; i < NUM_THREADS; i++)
    {
        tid[i] = i;
        DRIVER_API_CALL(hipCtxCreate(&(cuCtx[i]), 0, 0));
        DRIVER_API_CALL(hipCtxPopCurrent(&(cuCtx[i])));

        rc = pthread_create(&tidarr[i], NULL, thread_gpu, &(tid[i]));
        if(rc)
        {
            fprintf(stderr, "\n ERROR: return code from pthread_create is %d \n", rc);
            exit(1);
        }
        fprintf(stderr, "\n Main thread %lu. Created new thread (%lu) in iteration %d ...\n",
                (unsigned long)pthread_self(), (unsigned long)tidarr[i], i);
    }

    // Join all threads when complete
    for (i=0; i<NUM_THREADS; i++) {
        pthread_join(tidarr[i], NULL);
        fprintf(stderr, "IDX: %d: TID: %lu: Done! Joined main thread.\n", i, (unsigned long)tidarr[i]);
    }

    // Destroy all CUDA contexts for all threads/GPUs
    for (i=0; i<NUM_THREADS; i++) {
        DRIVER_API_CALL(hipCtxDestroy(cuCtx[i]));
    }
    printf("Main thread exit!\n");
    return 0;
}
